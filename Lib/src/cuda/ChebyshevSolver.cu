#include "hip/hip_runtime.h"
/* Copyright 2016 Kristofer Björnson
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file ChebyshevSolver.cu
 *
 *  @author Kristofer Björnson
 */

#include "ChebyshevSolver.h"
#include "GPUResourceManager.h"
#include "HALinkedList.h"
#include "Streams.h"
#include "TBTKMacros.h"

#include <hip/hip_complex.h>
#include <hipsparse.h>

#include <math.h>

using namespace std;

namespace TBTK{

complex<double> minus_one(-1., 0.);
complex<double> one(1., 0.);
complex<double> two(2., 0.);
complex<double> zero(0., 0.);
complex<double> i(0., 1.);

__global__
void extractCoefficients(
	hipDoubleComplex *jResult,
	int basisSize,
	hipDoubleComplex *coefficients,
	int currentCoefficient,
	int *coefficientMap,
	int numCoefficients
){
	int to = blockIdx.x*blockDim.x + threadIdx.x;
	if(to < basisSize && coefficientMap[to] != -1)
		coefficients[coefficientMap[to]*numCoefficients + currentCoefficient] = jResult[to];
}

void ChebyshevSolver::calculateCoefficientsGPU(
	Index to,
	Index from,
	complex<double> *coefficients,
	int numCoefficients,
	double broadening
){
	vector<Index> toVector;
	toVector.push_back(to);
	calculateCoefficientsGPU(toVector, from, coefficients, numCoefficients, broadening);
}

void ChebyshevSolver::calculateCoefficientsGPU(
	vector<Index> &to,
	Index from,
	complex<double> *coefficients,
	int numCoefficients,
	double broadening
){
	TBTKAssert(
		model != NULL,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"Model not set",
		"Use ChebyshevSolver::setModel() to set model."
	);
	TBTKAssert(
		scaleFactor > 0,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"Scale factor must be larger than zero.",
		"Use ChebyshevSolver::setScaleFactor() to set scale factor."
	);
	TBTKAssert(
		numCoefficients > 0,
		"ChebyshevSolver::calculateCoefficients()",
		"numCoefficients has to be larger than zero.",
		""
	);

//	int device = allocateDeviceGPU();
	int device = GPUResourceManager::getInstance().allocateDevice();

	TBTKAssert(
		hipSetDevice(device) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA set device error for device " << device << ".",
		""
	);

	AmplitudeSet *amplitudeSet = model->getAmplitudeSet();

	int fromBasisIndex = amplitudeSet->getBasisIndex(from);
	int *coefficientMap = new int[amplitudeSet->getBasisSize()];
	for(int n = 0; n < amplitudeSet->getBasisSize(); n++)
		coefficientMap[n] = -1;
	for(int n = 0; n < to.size(); n++)
		coefficientMap[amplitudeSet->getBasisIndex(to.at(n))] = n;

	if(isTalkative){
		Streams::out << "ChebyshevSolver::calculateCoefficientsGPU\n";
		Streams::out << "\tFrom Index: " << fromBasisIndex << "\n";
		Streams::out << "\tBasis size: " << amplitudeSet->getBasisSize() << "\n";
		Streams::out << "\tUsing damping: ";
		if(damping != NULL)
			Streams::out << "Yes\n";
		else
			Streams::out << "No\n";
	}

	complex<double> *jIn1 = new complex<double>[amplitudeSet->getBasisSize()];
	complex<double> *jIn2 = new complex<double>[amplitudeSet->getBasisSize()];
	complex<double> *jTemp = NULL;
	for(int n = 0; n < amplitudeSet->getBasisSize(); n++){
		jIn1[n] = 0.;
		jIn2[n] = 0.;
	}

	//Set up initial state (|j0>)
	jIn1[fromBasisIndex] = 1.;

	for(int n = 0; n < amplitudeSet->getBasisSize(); n++)
		if(coefficientMap[n] != -1)
			coefficients[coefficientMap[n]*numCoefficients] = jIn1[n];

	const int numHoppingAmplitudes = amplitudeSet->getNumMatrixElements();
	const int *cooHARowIndices_host = amplitudeSet->getCOORowIndices();
	const int *cooHAColIndices_host = amplitudeSet->getCOOColIndices();
	const complex<double> *cooHAValues_host = amplitudeSet->getCOOValues();

	//Initialize GPU
	complex<double> *jIn1_device;
	complex<double> *jIn2_device;
	int *cooHARowIndices_device;
	int *csrHARowIndices_device;
	int *cooHAColIndices_device;
	complex<double> *cooHAValues_device;
	complex<double> *coefficients_device;
	int *coefficientMap_device;
	complex<double> *damping_device = NULL;

	int totalMemoryRequirement = amplitudeSet->getBasisSize()*sizeof(complex<double>);
	totalMemoryRequirement += amplitudeSet->getBasisSize()*sizeof(complex<double>);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(int);
	totalMemoryRequirement += amplitudeSet->getBasisSize()*sizeof(int);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(int);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(complex<double>);
	totalMemoryRequirement += to.size()*numCoefficients*sizeof(complex<double>);
	totalMemoryRequirement += amplitudeSet->getBasisSize()*sizeof(int);
	if(damping != NULL)
		totalMemoryRequirement += amplitudeSet->getBasisSize()*sizeof(complex<double>);
	if(isTalkative){
		Streams::out << "\tCUDA memory requirement: ";
		if(totalMemoryRequirement < 1024)
			Streams::out << totalMemoryRequirement/1024 << "B\n";
		else if(totalMemoryRequirement < 1024*1024)
			Streams::out << totalMemoryRequirement/1024 << "KB\n";
		else
			Streams::out << totalMemoryRequirement/1024/1024 << "MB\n";
	}

	TBTKAssert(
		hipMalloc(
			(void**)&jIn1_device,
			amplitudeSet->getBasisSize()*sizeof(complex<double>)
		) == hipSuccess,
		"ChebyshevSOlver::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating jIn1_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&jIn2_device,
			amplitudeSet->getBasisSize()*sizeof(complex<double>)
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating jIn2_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&cooHARowIndices_device,
			numHoppingAmplitudes*sizeof(int)
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating cooHARowIndices_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&csrHARowIndices_device,
			(amplitudeSet->getBasisSize()+1)*sizeof(int)
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating csrHARowIndices_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&cooHAColIndices_device,
			numHoppingAmplitudes*sizeof(int)
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating cooHAColIndices_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&cooHAValues_device,
			numHoppingAmplitudes*sizeof(complex<double>)
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating cooHAValues_device.",
		""
	)
	TBTKAssert(
		hipMalloc(
			(void**)&coefficients_device,
			to.size()*numCoefficients*sizeof(complex<double>)
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating coefficients_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&coefficientMap_device,
			amplitudeSet->getBasisSize()*sizeof(int)
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating coefficientMap_device.",
		""
	);
	if(damping != NULL){
		TBTKAssert(
			hipMalloc(
				(void**)&damping_device,
				amplitudeSet->getBasisSize()*sizeof(complex<double>)
			) == hipSuccess,
			"ChebyshevSolver::calculateCoefficientsGPU()",
			"CUDA malloc error while allocating damping_device.",
			""
		);
	}

	TBTKAssert(
		hipMemcpy(
			jIn1_device,
			jIn1,
			amplitudeSet->getBasisSize()*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying jIn1.",
		""
	);
	TBTKAssert(
		hipMemcpy(
			jIn2_device,
			jIn2,
			amplitudeSet->getBasisSize()*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying jIn2.",
		""
	);
	TBTKAssert(
		hipMemcpy(
			cooHARowIndices_device,
			cooHARowIndices_host,
			numHoppingAmplitudes*sizeof(int),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying cooHARowIndices.",
		""
	);
	TBTKAssert(
		hipMemcpy(
			cooHAColIndices_device,
			cooHAColIndices_host,
			numHoppingAmplitudes*sizeof(int),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficients()",
		"CUDA memcpy error while copying cooHAColIndices.",
		""
	)
	TBTKAssert(
		hipMemcpy(
			cooHAValues_device,
			cooHAValues_host,
			numHoppingAmplitudes*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying cooHAValues.",
		""
	);
	TBTKAssert(
		hipMemcpy(
			coefficients_device,
			coefficients,
			to.size()*numCoefficients*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficients()",
		"CUDA memcpy error while copying coefficients.",
		""
	)
	TBTKAssert(
		hipMemcpy(
			coefficientMap_device,
			coefficientMap,
			amplitudeSet->getBasisSize()*sizeof(int),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying coefficientMap.",
		""
	);
	if(damping != NULL){
		TBTKAssert(
			hipMemcpy(
				damping_device,
				damping,
				amplitudeSet->getBasisSize()*sizeof(complex<double>),
				hipMemcpyHostToDevice
			) == hipSuccess,
			"ChebyshevSolver::calculateCoefficientsGPU()",
			"CUDA memcpy error while copying damping.",
			""
		);
	}

	hipsparseHandle_t handle = NULL;
	TBTKAssert(
		hipsparseCreate(&handle) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"cuSPARSE create error.",
		""
	);

	hipsparseMatDescr_t descr = NULL;
	TBTKAssert(
		hipsparseCreateMatDescr(&descr) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"cuSPARSE create matrix descriptor error.",
		""
	);

	TBTKAssert(
		hipsparseSetMatType(
			descr,
			HIPSPARSE_MATRIX_TYPE_GENERAL
		) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"cuSPARSE set matrix type error.",
		""
	);
	TBTKAssert(
		hipsparseSetMatIndexBase(
			descr,
			HIPSPARSE_INDEX_BASE_ZERO
		) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"cuSPARSE set matrix index base error.",
		""
	);

	TBTKAssert(
		hipsparseXcoo2csr(
			handle,
			cooHARowIndices_device,
			numHoppingAmplitudes,
			amplitudeSet->getBasisSize(),
			csrHARowIndices_device,
			HIPSPARSE_INDEX_BASE_ZERO
		) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"cuSPARSE COO to CSR error.",
		""
	);

	//Calculate |j1>
	int block_size = 1024;
	int num_blocks = amplitudeSet->getBasisSize()/block_size + (amplitudeSet->getBasisSize()%block_size == 0 ? 0:1);
	if(isTalkative){
		Streams::out << "\tCUDA Block size: " << block_size << "\n";
		Streams::out << "\tCUDA Num blocks: " << num_blocks << "\n";
	}

	complex<double> multiplier = one/scaleFactor;
	TBTKAssert(
		hipsparseZcsrmv(
			handle,
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			amplitudeSet->getBasisSize(),
			amplitudeSet->getBasisSize(),
			numHoppingAmplitudes,
			(hipDoubleComplex*)&multiplier,
			descr,
			(hipDoubleComplex*)cooHAValues_device,
			csrHARowIndices_device,
			cooHAColIndices_device,
			(hipDoubleComplex*)jIn1_device,
			(hipDoubleComplex*)&zero,
			(hipDoubleComplex*)jIn2_device
		) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevSolver::calculateCoefficentsGPU()",
		"Matrix-vector multiplication error.",
		""
	);

	extractCoefficients <<< num_blocks, block_size >>> ((hipDoubleComplex*)jIn2_device,
								amplitudeSet->getBasisSize(),
								(hipDoubleComplex*)coefficients_device,
								1,
								coefficientMap_device,
								numCoefficients);
	jTemp = jIn2_device;
	jIn2_device = jIn1_device;
	jIn1_device = jTemp;

	if(isTalkative)
		Streams::out << "\tProgress (100 coefficients per dot): ";

	//Iteratively calculate |jn> and corresponding Chebyshev coefficients.
	for(int n = 2; n < numCoefficients; n++){
		multiplier = two/scaleFactor;
		TBTKAssert(
			hipsparseZcsrmv(
				handle,
				HIPSPARSE_OPERATION_NON_TRANSPOSE,
				amplitudeSet->getBasisSize(),
				amplitudeSet->getBasisSize(),
				numHoppingAmplitudes,
				(hipDoubleComplex*)&multiplier,
				descr,
				(hipDoubleComplex*)cooHAValues_device,
				csrHARowIndices_device,
				cooHAColIndices_device,
				(hipDoubleComplex*)jIn1_device,
				(hipDoubleComplex*)&minus_one,
				(hipDoubleComplex*)jIn2_device
			) == HIPSPARSE_STATUS_SUCCESS,
			"ChebyshevSolver::calculateCoefficientsGPU()",
			"Matrix-vector multiplication error.",
			""
		);

		extractCoefficients <<< num_blocks, block_size >>> ((hipDoubleComplex*)jIn2_device,
									amplitudeSet->getBasisSize(),
									(hipDoubleComplex*)coefficients_device,
									n,
									coefficientMap_device,
									numCoefficients);

		jTemp = jIn2_device;
		jIn2_device = jIn1_device;
		jIn1_device = jTemp;

		if(isTalkative){
			if(n%100 == 0)
				Streams::out << "." << flush;
			if(n%1000 == 0)
				Streams::out << " " << flush;
		}
	}
	if(isTalkative)
		Streams::out << "\n";

	TBTKAssert(
		hipMemcpy(
			coefficients,
			coefficients_device,
			to.size()*numCoefficients*sizeof(complex<double>),
			hipMemcpyDeviceToHost
		) == hipSuccess,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying coefficients.",
		""
	);

	TBTKAssert(
		hipsparseDestroyMatDescr(descr) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"cuSPARSE destroy matrix descriptor error.",
		""
	);
	descr = NULL;

	TBTKAssert(
		hipsparseDestroy(handle) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevSolver::calculateCoefficientsGPU()",
		"cuSPARSE destroy error.",
		""
	);
	handle = NULL;

	delete [] jIn1;
	delete [] jIn2;
	delete [] coefficientMap;

	hipFree(jIn1_device);
	hipFree(jIn2_device);
	hipFree(cooHARowIndices_device);
	hipFree(csrHARowIndices_device);
	hipFree(cooHAColIndices_device);
	hipFree(cooHAValues_device);
	hipFree(coefficients_device);
	hipFree(coefficientMap_device);
	if(damping != NULL)
		hipFree(damping_device);

//	freeDeviceGPU(device);
	GPUResourceManager::getInstance().freeDevice(device);

	//Lorentzian convolution
	double lambda = broadening*numCoefficients;
	for(int n = 0; n < numCoefficients; n++)
		for(int c = 0; c < to.size(); c++)
			coefficients[n + c*numCoefficients] = coefficients[n + c*numCoefficients]*sinh(lambda*(1 - n/(double)numCoefficients))/sinh(lambda);
}

__global__
void calculateGreensFunction(
	hipDoubleComplex *greensFunction,
	hipDoubleComplex *coefficients,
	hipDoubleComplex *lookupTable,
	int numCoefficients,
	int energyResolution
){
	int e = blockIdx.x*blockDim.x + threadIdx.x;
	if(e < energyResolution)
		for(int n = 0; n < numCoefficients; n++)
			greensFunction[e] = hipCadd(greensFunction[e], hipCmul(lookupTable[n*energyResolution + e], coefficients[n]));
//			greensFunction[e] += lookupTable[n*energyResolution + e]*coefficients[n];
}

void ChebyshevSolver::loadLookupTableGPU(){
	if(isTalkative)
		Streams::out << "CheyshevSolver::loadLookupTableGPU\n";

	TBTKAssert(
		generatingFunctionLookupTable != NULL,
		"ChebyshevSolver::loadLookupTableGPU()",
		"Lookup table has not been generated.",
		"Call ChebyshevSolver::generateLokupTable() to generate lookup table."
	);
	if(generatingFunctionLookupTable_device != NULL)
		destroyLookupTableGPU();
/*	TBTKAssert(
		generatingFunctionLookupTable_device == NULL,
		"ChebyshevSolver::loadLookupTableGPU()",
		"Lookup table already loaded.",
		""
	);*/

	complex<double> *generatingFunctionLookupTable_host = new complex<double>[lookupTableNumCoefficients*lookupTableResolution];
	for(int n = 0; n < lookupTableNumCoefficients; n++)
		for(int e = 0; e < lookupTableResolution; e++)
			generatingFunctionLookupTable_host[n*lookupTableResolution + e] = generatingFunctionLookupTable[n][e];

	int memoryRequirement = lookupTableNumCoefficients*lookupTableResolution*sizeof(complex<double>);
	if(isTalkative){
		Streams::out << "\tCUDA memory requirement: ";
		if(memoryRequirement < 1024)
			Streams::out << memoryRequirement << "B\n";
		else if(memoryRequirement < 1024*1024)
			Streams::out << memoryRequirement/1024 << "KB\n";
		else
			Streams::out << memoryRequirement/1024/1024 << "MB\n";
	}

//	generatingFunctionLookupTable_device = new complex<double>**[numDevices];
	generatingFunctionLookupTable_device = new complex<double>**[GPUResourceManager::getInstance().getNumDevices()];

//	for(int n = 0; n < numDevices; n++){
	for(int n = 0; n < GPUResourceManager::getInstance().getNumDevices(); n++){
		TBTKAssert(
			hipSetDevice(n) == hipSuccess,
			"ChebyshevSolver::loadLookupTableGPU()",
			"CUDA set device error for device " << n << ".",
			""
		);

		TBTKAssert(
			hipMalloc(
				(void**)&generatingFunctionLookupTable_device[n],
				lookupTableNumCoefficients*lookupTableResolution*sizeof(complex<double>)
			)  == hipSuccess,
			"ChebyshevSolver::loadLookupTableGPU()",
			"CUDA malloc error while allocating generatingFunctionLookupTable_device.",
			""
		);

		TBTKAssert(
			hipMemcpy(
				generatingFunctionLookupTable_device[n],
				generatingFunctionLookupTable_host,
				lookupTableNumCoefficients*lookupTableResolution*sizeof(complex<double>),
				hipMemcpyHostToDevice
			) == hipSuccess,
			"ChebyshevSolver::loadLookupTableGPU()",
			"CUDA memcpy error while copying generatingFunctionLookupTable_device.",
			""
		);
	}

	delete [] generatingFunctionLookupTable_host;
}

void ChebyshevSolver::destroyLookupTableGPU(){
	if(isTalkative)
		Streams::out << "ChebyshevSolver::destroyLookupTableGPU\n";

	TBTKAssert(
		generatingFunctionLookupTable_device != NULL,
		"ChebyshevSolver::destroyLookupTableGPU()",
		"No lookup table loaded onto GPU.\n",
		""
	);

//	for(int n = 0; n < numDevices; n++){
	for(int n = 0; n < GPUResourceManager::getInstance().getNumDevices(); n++){
		hipFree(generatingFunctionLookupTable_device[n]);
	}

	delete [] generatingFunctionLookupTable_device;
	generatingFunctionLookupTable_device = NULL;
}

void ChebyshevSolver::generateGreensFunctionGPU(
	complex<double> *greensFunction,
	complex<double> *coefficients,
	GreensFunctionType type
){
//	int device = allocateDeviceGPU();
	int device = GPUResourceManager::getInstance().allocateDevice();

	TBTKAssert(
		hipSetDevice(device) == hipSuccess,
		"ChebyshevSolver::generateGreensFunctionGPU()",
		"CUDA set device error for device " << device << ".",
		""
	);

	if(isTalkative)
		Streams::out << "ChebyshevSolver::generateGreensFunctionGPU\n";

	TBTKAssert(
		generatingFunctionLookupTable_device != NULL,
		"ChebyshevSolver::generateGreensFunctionGPU()",
		"No lookup table loaded onto GPU.",
		""
	);
	TBTKAssert(
		type == GreensFunctionType::Retarded,
		"ChebyshevSolver::generateGreensFunctionGPU()",
		"Only evaluation of retarded Green's function is implemented for GPU so far.",
		"Use CPU evaluation instead."
	);

	for(int e = 0; e < lookupTableResolution; e++)
		greensFunction[e] = 0.;

	complex<double> *greensFunction_device;
	complex<double> *coefficients_device;

	TBTKAssert(
		hipMalloc(
			(void**)&greensFunction_device,
			lookupTableResolution*sizeof(complex<double>)
		)  == hipSuccess,
		"ChebyshevSolver::generateGreensFunctionGPU()",
		"CUDA malloc error while allocating greensFunction_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&coefficients_device,
			lookupTableNumCoefficients*sizeof(complex<double>)
		)  == hipSuccess,
		"ChebyshevSolver::generateGreensFunctionGPU()",
		"CUDA malloc error while allocating coefficients_device.",
		""
	);

	TBTKAssert(
		hipMemcpy(
			greensFunction_device,
			greensFunction,
			lookupTableResolution*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevSolver::generateGreensFunctionGPU()",
		"CUDA memcpy error while copying greensFunction.",
		""
	);
	TBTKAssert(
		hipMemcpy(
			coefficients_device,
			coefficients,
			lookupTableNumCoefficients*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevSolver::generateGreensFunctionGPU()",
		"CUDA memcpy error while copying coefficients.",
		""
	);

	int block_size = 1024;
	int num_blocks = lookupTableResolution/block_size + (lookupTableResolution%block_size == 0 ? 0:1);

	if(isTalkative){
		Streams::out << "\tCUDA Block size: " << block_size << "\n";
		Streams::out << "\tCUDA Num blocks: " << num_blocks << "\n";
	}

	calculateGreensFunction <<< num_blocks, block_size>>> ((hipDoubleComplex*)greensFunction_device,
								(hipDoubleComplex*)coefficients_device,
								(hipDoubleComplex*)generatingFunctionLookupTable_device[device],
								lookupTableNumCoefficients,
								lookupTableResolution);

	TBTKAssert(
		hipMemcpy(
			greensFunction,
			greensFunction_device,
			lookupTableResolution*sizeof(complex<double>),
			hipMemcpyDeviceToHost
		) == hipSuccess,
		"ChebyshevSolver::generateGreensFunctionGPU()",
		"CUDA memcpy error while copying greensFunction_device.",
		""
	);

	hipFree(greensFunction_device);
	hipFree(coefficients_device);

//	freeDeviceGPU(device);
	GPUResourceManager::getInstance().freeDevice(device);
}

/*void ChebyshevSolver::createDeviceTableGPU(){
	hipGetDeviceCount(&numDevices);

	Streams::out << "Num GPU devices: " << numDevices << "\n";

	if(numDevices > 0){
		busyDevices = new bool[numDevices];
		for(int n = 0; n < numDevices; n++)
			busyDevices[n] = false;
	}
}

void ChebyshevSolver::destroyDeviceTableGPU(){
	if(numDevices > 0)
		delete [] busyDevices;
}

int ChebyshevSolver::allocateDeviceGPU(){
	int device = 0;
	bool done = false;
	while(!done){
		omp_set_lock(&busyDevicesLock);
		#pragma omp flush
		{
			for(int n = 0; n < numDevices; n++){
				if(!busyDevices[n]){
					device = n;
					busyDevices[n] = true;
					done = true;
					break;
				}
			}
		}
		#pragma omp flush
		omp_unset_lock(&busyDevicesLock);
	}

	return device;
}

void ChebyshevSolver::freeDeviceGPU(int device){
	omp_set_lock(&busyDevicesLock);
	#pragma omp flush
	{
		busyDevices[device] = false;
	}
	#pragma omp flush
	omp_unset_lock(&busyDevicesLock);
}*/

};	//End of namespace TBTK
